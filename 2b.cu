

#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>

__global__ void mul(int *a, int *b, int *c, int n)
{
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int sum = 0;
	int i;
	if(row < n && col < n)
	{
		for(i =0; i<n; i++)
		{
			sum += a[row*n+i]*b[i*n+col];
		}
		c[row*n+col] =  sum;
	}
	
}

int main(int argc, char **argv)
{
	int N;
	int *a, *b, *c, *d, *da, *db, *dc;
	int i,j,k;
	scanf("%d",&N);
	
	
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	a = (int *)calloc(N*N,sizeof(int));
	b = (int *)calloc(N*N,sizeof(int));
	c = (int *)calloc(N*N,sizeof(int));
	d = (int *)calloc(N*N,sizeof(int));

	for(i = 0; i < N*N; i++)
	{
		a[i] = rand()%48;
		b[i] = rand()%50;
	}
	
	int size = N*N*sizeof(int);

	hipMalloc(&da,size);
	hipMalloc(&db,size);
	hipMalloc(&dc,size);
	
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	
	dim3 grid((N+15)/16,(N+15)/16);
	dim3 block(16,16);
	
	hipEventRecord(start);
	mul<<<grid,block>>>(da,db,dc,N);
	hipEventRecord(stop);
	
	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float ms;
	hipEventElapsedTime(&ms,start,stop);
	
	for(i = 0; i<N; i++)
		for(j = 0; j<N; j++)
		{
			d[i*N+j] = 0.0;
			for(k = 0; k<N; k++)
			{
							d[i*N+j] += a[i*N+k]*b[k*N+j];
			}
			if(d[i*N+j] != c[i*N+j])
			{
				printf("Inmcoreect\n");
				exit(-2);
			}
		}
		
	printf("%lf\n",ms);	
}