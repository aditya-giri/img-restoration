/* Compute the sum of two vectors using CUDA
 * Vishwas S
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
__global__ void add(int *a, int *b, int *c, int n)
{
	int id = blockIdx.x*blockDim.x + threadIdx.x;
	if(id<n)
		c[id] = a[id] + b[id];
}

int main()
{
	int N;
	int *a, *b, *c, *da, *db, *dc;
	
	scanf("%d",&N);
	
	
	hipEvent_t start, stop;
	
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	a = (int *)calloc(N,sizeof(int));
	b = (int *)calloc(N,sizeof(int));
	c = (int *)calloc(N,sizeof(int));
	
	for(int i = 0; i < N; i++)
	{
		a[i] = rand()%48;
		b[i] = rand()%50;
	}
	
	int size = N*sizeof(int);

	hipMalloc(&da,size);
	hipMalloc(&db,size);
	hipMalloc(&dc,size);
	
	hipMemcpy(da,a,size,hipMemcpyHostToDevice);
	hipMemcpy(db,b,size,hipMemcpyHostToDevice);
	
	
	hipEventRecord(start);
	add<<<(N+511)/512,512>>>(da,db,dc,N); //block count, threads per block
	hipEventRecord(stop);
	
	hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);
	hipEventSynchronize(stop);
	float ms;
	
	hipEventElapsedTime(&ms,start,stop);
	printf("%f\n",ms);
}